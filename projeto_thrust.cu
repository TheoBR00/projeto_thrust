#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <vector>
#include <numeric>
#include <algorithm>
#include <random>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/functional.h>
#include <thrust/transform.h>
#include <thrust/random.h>
#include <thrust/shuffle.h>
#include <thrust/fill.h>
#include <thrust/execution_policy.h>

using namespace std;

struct graph{
    int id;
    float x;
    float y;
    bool vis;
};

struct ponto{
    int id;
    float x;
    float y;
    //float d;
};

struct dist_calc
{
  graph *vetor;
  dist_calc(graph *vetor_) : vetor(vetor_) {};
  __device__
  float operator()(int ind){
    return sqrt(pow(static_cast<graph>(vetor[ind]).x - vetor[ind+1].x, 2) + pow(static_cast<graph>(vetor[ind]).y - vetor[ind+1].y, 2));
  }
};


struct swap_vec
{
  graph *vetor;
  unsigned int n;
  swap_vec(graph *vetor_n, unsigned int n) : vetor(vetor_n), n(n) {};
  __device__
  float operator()(int posi){

    float dist = 10000000;

    float d;

    //return swap(static_cast<graph>(vetor[posi]), static_cast<graph>(vetor[posi+1]));

    for(int ct = 0; ct < posi; ct++){

      for(int ct_2 = 0; ct_2 < n-1; ct_2++){
        graph p1 = vetor[ct_2];
        graph p2 = vetor[ct_2+1];
        vetor[ct_2] = p2;
        vetor[ct_2+1] = p1;

        d = sqrt(pow(vetor[ct_2].x - vetor[ct_2+1].x, 2) + pow(vetor[ct_2].y - vetor[ct_2+1].y, 2));

        if(d < dist){
          dist = d;
        }
      }
      return d;
    }

  }
};

//float __device__ dist_calc(int ind){

//  return sqrt(pow(raw_cast<graph>(p1).x - p2.x, 2) + pow(raw_cast<graph>(p1).y - p2.y, 2));

//}


int main(){

    std::cout << "Main" << std::endl;

    ofstream myfile;

    std::cout << "ofstream myfile" << std::endl;

    unsigned int n;

    

    float x0;
    float y0;
    //float x1;
    //float y1;
    
    std::cin >> n;

    std::cout << "Carregou n: " << n << std::endl;

    //float dist;

    //std::vector<int>vec_ids;

    thrust::device_vector<int> vec_ids(n);

    std::cout << "Criou vec_ids" << std::endl;

    int chegou = 0;

    //if(n - 1 == 1){
    //    std::cin >> x1;
    //    std::cin >> y1;

    //    dist = sqrt(pow(x0 - x1, 2) + pow(y0 - y1, 2));
    //    return dist;
    //}

    bool status = true;

    //std::vector<graph> vec_pontos;

    thrust::device_vector<graph> vec_pontos(n);

    int i = 0;
    int pos = 0;

    thrust::device_vector<graph> array[10*n];

    thrust::device_vector<float> array_float[10*n];

    for(int r = 0; r < n; r++){
        std::cin >> x0;
        std::cin >> y0;

        graph g;
        g.id = r;
        g.x = x0;
        g.y = y0;
        g.vis = false;

        vec_pontos[r] = g;

    }

    for(int b = 0; b < n; b++){
        //std::cout << "id: " << vec_pontos[b].id << endl;
    }

    int num = 0;
    int count = 0;
    int sw = 0;

    float d_total = 0.0;

    //std::default_random_engine e(seed);
    thrust::random::default_random_engine generator;
    thrust::uniform_int_distribution<int> distribution(1,10);

    float d_a = 10000000.0;


    thrust::device_vector<graph> vec(n);

    //std::vector<graph> vec;
    

    for(int rodada = 0; rodada < 10; rodada++){
        auto sorteia = distribution(generator);
        std::cout << "random: " << sorteia << endl;

        std::cout << "Antes do shuffle" << std::endl;

        for(int vetores = 0; vetores < 10*n; vetores++){

          array[vetores] = thrust::device_vector<graph>(n);

          for(int sr = 0; sr < n; sr++){
            array[vetores][sr] = vec_pontos[sr];
          }

          thrust::shuffle(thrust::device, array[vetores].begin(), array[vetores].end(), generator);

          thrust::shuffle(thrust::device, vec_pontos.begin(), vec_pontos.end(), generator);

        }

        for(int lm = 0; lm < 10*n; lm++){
          std::cout << " " << std::endl;
          for(int k = 0; k < n; k++){
            std::cout << "VETORES[lm" << "] " << "= " << static_cast<graph>(array[lm][k]).id << std::endl;

            }
          }

          std::cout << " " << std::endl;

        //thrust::shuffle(thrust::device, vec_pontos.begin(), vec_pontos.end(), generator);

        std::cout << "Depois do shuffle" << std::endl;

        //std::shuffle(vec_pontos.begin(), vec_pontos.end(), default_random_engine(sorteia));
        status = true;

        count = 0;

        chegou = 0;
    
        float d_to = 0;

        //for(int contagem = 0; contagem < n; contagem++){
            //std::cout << vec_pontos[contagem].x << " " << vec_pontos[contagem].y << endl;
        //}

        while(status){

            if(count == n-1){
                status = false;
            }

            d_to = 0;

            //std::cout << "d_to = " << d_to << std::endl;

            //std::cout << "Reiniciou" << endl;

            thrust::device_vector<float> vec_dists(n);

            for(i = 0; i < n; i++){

                std::cout << "Pegou pontos" << std::endl;
                
                if(static_cast<graph>(vec_pontos[i]).vis == false && i < n-1) {

                    thrust::counting_iterator<int> iter(0);

                    thrust::transform(iter, iter+(n-1), vec_pontos.begin(), dist_calc(thrust::raw_pointer_cast(vec_pontos.data())));

                    for(int tam = 0; tam < n; tam++){
                      std::cout << "vec_dists = " << vec_dists[tam] << std::endl;
                    }

                    //float d_novo = sqrt(pow(static_cast<graph>(vec_pontos[i]).x - p1.x, 2) + pow(static_cast<graph>(vec_pontos[i]).y - p1.y, 2));
                    //d_to += d_novo;
                    //std::cout << "d_novo: " << d_novo << endl;
                    //std::cout << vec_pontos[i].x << vec_pontos[i].y << " " << p1.x << p1.y << endl;
                    std::cout << "i: " << i << endl;

                    //if(d_novo < d_a){
                    //    d_a = d_novo;
                        //num = i;
                    //}

                }

                if(i == n-1){
                  std::cout << "i == n - 1" << std::endl;
                  //d_to += sqrt(pow(static_cast<graph>(vec_pontos[i]).x - static_cast<graph>(vec_pontos[0]).x, 2) + pow(static_cast<graph>(vec_pontos[i]).y - static_cast<graph>(vec_pontos[0]).y, 2));
                  //std::cout << "i: " << i << endl;
                  std::cout << "d_to: " << d_to << endl;
                }

                //d_to = d_novo;



            }

            std::cout << "Percorreu for" << std::endl;

            if(d_to <= d_a){
                thrust::device_vector<graph> vec(n);
                d_a = d_to;
                for(int u = 0; u < n; u++){
                    //vec.push_back(vec_pontos[u]);
                    vec[u] = vec_pontos[u];
                }
                //d_total += d_a;
                //std::cout << "d_total: " << d_total << endl;
            }
            
            //vec_ids.push_back(vec_pontos[pos].id);
            std::cout << "Antes do static cast" << std::endl;
            vec_ids[count] = static_cast<graph>(vec_pontos[pos]).id;
            std::cout << "Fez static cast" << std::endl;



            //vec_pontos[pos].vis = true;

            //if(count == n-1){
            //    swap(vec_pontos[count], vec_pontos[0]);
            //}

            if(count < n-1){

                //swap(vec_pontos[count], vec_pontos[count+1]);

                thrust::counting_iterator<float> iterador(0);

                thrust::transform(iterador, iterador+(n-1), array_float.begin(), swap_vec(thrust::raw_pointer_cast(array.data())));
                for(int it = 0; it < n; it++){
                  std::cout << "swap vec_pontos: " << static_cast<graph>(vec_pontos[it]).id << std::endl;
                }
            }
            
                

            for(int sw = 0; sw < n; sw++){
                //std::cout << "swap: " << vec_pontos[sw].x << " " << vec_pontos[sw].y << endl;
            }

            std::cerr << "local: "  << d_to << " ";
            for(int erros = 0; erros < n; erros++){
                //std::cout << vec_pontos[erros].x << " " << vec_pontos[erros].y << endl;
                std::cerr << static_cast<graph>(vec_pontos[erros]).id << " ";
            }
            std::cerr << endl;

            count++;

            //pos = num;

            //if(count == n){
            //    float d_f = sqrt(pow(vec_pontos[pos].x - vec_pontos[0].x, 2) + pow(vec_pontos[pos].y - vec_pontos[0].y, 2));
            //    std::cout << "d_f: " << d_f << endl;
            //    d_to += d_f;

            //    std::cout << "d: "<< d_to << " " << 0 << endl;

            //    std::cout << endl;

            //    if(chegou == n-1){
            //        std::cout << "Trocou final" << endl;
            //        swap(vec_pontos[chegou], vec_pontos[0]);

            //        for(int posi = 0; posi < n; posi++){
            //            std::cout << "swap: " <<vec_pontos[posi].x << " " << vec_pontos[posi].y << endl;
            //        }

            //        std::cout << "Terminou swap" << endl;

            //        chegou = 0;
            //        sw = 0;
            //        status = false;
            //    }

            //    else{
            //        for(int vef = 0; vef < n; vef++){
            //            vec_pontos[vef].vis = false;
            //        }
            //        if(chegou < n-1){
            //            swap(vec_pontos[chegou], vec_pontos[chegou+1]);
            //        }
                    
            //        std::cout << "Chegou: " << chegou << endl;

            //        for(sw = 0; sw < n; sw++){
            //            std::cout << "swap: " <<vec_pontos[sw].x << " " << vec_pontos[sw].y << endl;
            //        }
            //        pos = 0;
            //        num = 0;

            //    }
                
            //    chegou++;

            //    count = 0;

            //    d_total = 0;

            //    std::vector<int>vec_ids;
                
            //}


            //std::vector<ponto>vec;

        }
    }

    std::cout << d_a << " " << 0 << endl;

    for(int min = 0; min < n; min++){
        std::cout << static_cast<graph>(vec[min]).id << " ";
    }
    std::cout << endl;
    
}